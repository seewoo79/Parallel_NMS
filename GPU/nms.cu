#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <stdbool.h>
using namespace cv;
using namespace std;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      
	  if(abort)
	  	exit(code);
   }
}


typedef struct
{
	float x, y, w, h, s;
}box;

__device__
float IOUcalc(box b1, box b2)
{
	//Get Union of boxes
	float tlx_u = min(b1.x, b2.x);
	float tly_u = min(b1.y, b2.y);

	float brx_u = max(b1.x + b1.w, b2.x + b2.w);
	float bry_u = max(b1.y + b1.h, b2.y + b2.h);

	//Get Intersection of boxes
	float tlx_i = max(b1.x, b2.x);
	float tly_i = max(b1.y, b2.y);

	float brx_i = min(b1.x + b1.w, b2.x + b2.w);
	float bry_i = min(b1.y + b1.h, b2.y + b2.h);

	float w_u = brx_u - tlx_u;
	float h_u = bry_u - tly_u;

	float w_i = brx_i - tlx_i;
	float h_i = bry_i - tly_i;

	float inter = (w_i * h_i) / (w_u * h_u);

	return inter;
}

__global__
void NMS_GPU(box *d_b, bool *d_res, const float theta)
{
	int target = blockIdx.x;
	int current = threadIdx.x;

	if(d_b[target].s > d_b[current].s)
	{
		float iou = IOUcalc(d_b[target], d_b[current]);
		if (iou > theta)	
		{
			d_res[current] = false; 
		}
	}
}


int main()
{
	int count = 6;
	Mat input = imread("./0.jpg",1);
	imshow("Input", input);
	waitKey(0);
	
	bool *h_res =(bool *)malloc(sizeof(bool)*count);
	
	for(int i = 0; i < count; i++)
		h_res[i] = true;
	
	box b[count];
	b[5].x = 155; b[5].y = 30; b[5].w = 70; b[5].h = 138; b[5].s = 0.5355;
	b[4].x = 150; b[4].y = 25; b[4].w = 74; b[4].h = 148; b[4].s = 0.2355;
	b[3].x = 11; b[3].y = 6; b[3].w = 74; b[3].h = 148; b[3].s = 0.42355;
	b[2].x = 12; b[2].y = 14; b[2].w = 70; b[2].h = 141; b[2].s = 0.60434;
	b[1].x = 16; b[1].y = 12; b[1].w = 64; b[1].h = 128; b[1].s = 0.79062;
	b[0].x = 11; b[0].y = 6; b[0].w = 74; b[0].h = 148; b[0].s = 0.11855;
	
	Mat temp = input.clone();
	for(int i = 0; i < count ; i++)
		rectangle(temp, Point(b[i].x,b[i].y), Point(b[i].x + b[i].w,b[i].y + b[i].h), Scalar(0,255,0), 1, 8, 0);
	imshow("Temp", temp);
	waitKey(0);

	box *d_b;
	bool *d_res;
	
	gpuErrchk(hipMalloc((void**)&d_res, count*sizeof(bool)));
	gpuErrchk(hipMemcpy(d_res, h_res,sizeof(bool) * count, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc((void**)&d_b, sizeof(box) * count));
	gpuErrchk(hipMemcpy(d_b, b, sizeof(box) * count, hipMemcpyHostToDevice));
		
	NMS_GPU <<< count, count >>> (d_b, d_res, 0.6f);
	
	hipDeviceSynchronize();
	
	gpuErrchk(hipMemcpy(h_res, d_res, sizeof(bool) * count, hipMemcpyDeviceToHost));
	
	for(int i = 0; i < count ; i++)
	{
		printf("res : %d\n", h_res[i]);
		if(*(h_res + i) == true)
		{
			printf("Results= %d--%d ",i,*(h_res+i));
			rectangle(input, Point(b[i].x,b[i].y), Point(b[i].x + b[i].w,b[i].y + b[i].h), Scalar(255,0,0), 1, 8, 0);
		}
	}

	imshow("Output",input);
	waitKey(0);
	return 0;
}
